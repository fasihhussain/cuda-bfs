#include <iostream>
#include <vector>
#include <queue>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

inline hipError_t checkCudaErr(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
    }
    return err;
}

void cpuBFS(int node, int nodes, int *adjacencyList, int *edgesOffset)
{
    queue<int> output_queue;
    vector<bool> visited(nodes, false);
    queue<int> temp_queue;

    output_queue.push(node);
    temp_queue.push(node);
    visited[node] = true;

    while (!temp_queue.empty())
    {
        int u = temp_queue.front();
        temp_queue.pop();
        for (int j = edgesOffset[u]; j < edgesOffset[u + 1]; j++)
        {
            int v = adjacencyList[j];
            if (!visited[v])
            {
                // printf("%d\n", v);
                output_queue.push(v);
                temp_queue.push(v);
                visited[v] = true;
            }
        }
    }

    // printf("CPU RESULT: %d\n", output_queue.size());
    // while (!output_queue.empty())
    // {
    //     printf("%d ", output_queue.front());
    //     output_queue.pop();
    // }
    // printf("\n");
}

__global__ void cudaBFS(int node, int nodes, int *adjacencyList, int *edgesOffset, int *currentQueue, int *nextQueue, int *visited)
{
    int thid = threadIdx.x;
    extern __shared__ int queues[];

    int *currentQueueSize = &queues[0];
    int *nextQueueSize = currentQueueSize + 1;

    if (thid == 0)
    {
        // printf("ADJ LIST FROM GPU:\n");
        // for (int i = 0; i < edgesOffset[nodes]; i++)
        // {
        //     printf("%d ", adjacencyList[i]);
        // }
        // printf("\n\n");

        // printf("OFFSET FROM GPU:\n");
        // for (int i = 0; i < nodes + 1; i++)
        // {
        //     printf("%d ", edgesOffset[i]);
        // }
        // printf("\n\n");

        currentQueue[0] = node;
        visited[node] = 1;
        *currentQueueSize = 1;
        *nextQueueSize = 0;

        // printf("QUEUE SIZE: %d\n", *currentQueueSize);
        // for (int i = 0; i < *currentQueueSize; i++)
        // {
        //     printf("%d ", currentQueue[i]);
        // }
        // printf("\n");
    }

    __syncthreads();

    while (*currentQueueSize > 0)
    {
        for (int i = thid; i < *currentQueueSize; i += blockDim.x)
        {
            int u = currentQueue[i];
            // printf("VISITED[%d] = %d\n", u, visited[u]);
            for (int j = edgesOffset[u]; j < edgesOffset[u + 1]; j++)
            {
                int v = adjacencyList[j];
                // printf("u = %d, v = %d, VISITED = %d\n", u, v, visited[v]);
                if (visited[v] == 0)
                {
                    visited[v] = 1;
                    int position = atomicAdd(nextQueueSize, 1);
                    nextQueue[position] = v;
                }
            }
        }
        __syncthreads();

        // if (thid == 0)
        // {
        //     printf("NEXT QUEUE SIZE: %d\n", *nextQueueSize);
        //     for (int i = 0; i < *nextQueueSize; i++)
        //     {
        //         printf("%d ", nextQueue[i]);
        //     }
        //     printf("\n\n\n");
        // }
        int *temp = currentQueue;
        currentQueue = nextQueue;
        nextQueue = temp;

        temp = currentQueueSize;
        currentQueueSize = nextQueueSize;
        nextQueueSize = temp;

        if (thid == 0)
        {
            *nextQueueSize = 0;
        }
        __syncthreads();
    }
}

int main(void)
{
    vector<int> v_nodes = {10, 100, 500};
    // , 1000, 5000, 10000, 20000};
    vector<double>
        v_prob = {0.05, 0.1, 0.3, 0.5};

    for (double p : v_prob)
    {
        printf("PROBABILITY: %f\n", p);
        for (int nodes : v_nodes)
        {

            vector<int> adjacencyList;
            int *h_edgesOffset = new int[nodes + 1];

            for (int i = 0; i < nodes; i++)
            {
                h_edgesOffset[i] = adjacencyList.size();
                for (int j = 0; j < nodes; j++)
                {
                    if ((float)rand() / RAND_MAX < p)
                    {
                        if (i != j)
                        {
                            adjacencyList.push_back(j);
                        }
                    }
                }
            }
            h_edgesOffset[nodes] = adjacencyList.size();

            int *h_adjacencyList = new int[adjacencyList.size()];
            for (int i = 0; i < adjacencyList.size(); i++)
            {
                h_adjacencyList[i] = adjacencyList[i];
            }

            int *h_visited = new int[nodes];
            int *d_adjacencyList, *d_edgesOffset, *d_currentQueue, *d_nextQueue, *d_visited;

            // printf("ADJ LIST:\n");
            // for (int i = 0; i < adjacencyList.size(); i++)
            // {
            //     printf("%d ", h_adjacencyList[i]);
            // }
            // printf("\n\n");

            // printf("OFFSET:\n");
            // for (int i = 0; i < nodes + 1; i++)
            // {
            //     printf("%d ", h_edgesOffset[i]);
            // }
            // printf("\n\n");

            checkCudaErr(hipMalloc((void **)&d_adjacencyList, adjacencyList.size() * sizeof(int)), "hipMalloc d_adjacencyList");
            checkCudaErr(hipMalloc((void **)&d_edgesOffset, (nodes + 1) * sizeof(int)), "hipMalloc d_edgesOffset");
            checkCudaErr(hipMalloc((void **)&d_currentQueue, nodes * sizeof(int)), "hipMalloc d_currentQueue");
            checkCudaErr(hipMalloc((void **)&d_nextQueue, nodes * sizeof(int)), "hipMalloc d_nextQueue");
            checkCudaErr(hipMalloc((void **)&d_visited, nodes * sizeof(int)), "hipMalloc d_visited");

            checkCudaErr(hipMemcpy(d_adjacencyList, h_adjacencyList, adjacencyList.size() * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy d_adjacencyList");
            checkCudaErr(hipMemcpy(d_edgesOffset, h_edgesOffset, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy d_edgesOffset");

            int NUMBER_OF_BLOCKS = 128;
            int NUMBER_OF_THREADS = 32;
            int SHARED_MEMORY_SIZE = 2 * sizeof(int);

            clock_t start, end;

            double time_taken_cpu = 0;
            double time_taken_gpu = 0;

            double t_cpu, t_gpu;

            int SAMPLES = 10;

            for (int node = 0; node < SAMPLES; node++)
            {
                start = clock();
                cpuBFS(node, nodes, h_adjacencyList, h_edgesOffset);
                end = clock();

                t_cpu = ((double)(end - start)) * 1000 / CLOCKS_PER_SEC;
                time_taken_cpu += t_cpu;

                hipMemset(d_visited, 0, nodes * sizeof(int));
                start = clock();
                cudaBFS<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS, SHARED_MEMORY_SIZE>>>(node, nodes, d_adjacencyList, d_edgesOffset, d_currentQueue, d_nextQueue, d_visited);
                checkCudaErr(hipDeviceSynchronize(), "device synchronize");
                end = clock();

                t_gpu = ((double)(end - start)) * 1000 / CLOCKS_PER_SEC;
                time_taken_gpu += t_gpu;

                hipMemcpy(h_visited, d_visited, nodes * sizeof(int), hipMemcpyDeviceToHost);

                // printf("GPU RESULT:\n");
                // for (int i = 0; i < nodes; i++)
                // {
                //     if (h_visited[i] == 1)
                //     {
                //         printf("%d ", i);
                //     }
                // }
                // printf("\n\n");
            }
            time_taken_cpu /= SAMPLES;
            time_taken_gpu /= SAMPLES;
            printf("NODES: %d\t\tCPU: %f ms\t\tGPU: %f ms\n", nodes, time_taken_cpu, time_taken_gpu);

            free(h_adjacencyList);
            free(h_edgesOffset);
            free(h_visited);

            checkCudaErr(hipFree(d_adjacencyList), "hipFree d_adjacencyList");
            checkCudaErr(hipFree(d_edgesOffset), "hipFree d_edgesOffset");
            checkCudaErr(hipFree(d_currentQueue), "hipFree d_currentQueue");
            checkCudaErr(hipFree(d_nextQueue), "hipFree d_nextQueue");
            checkCudaErr(hipFree(d_visited), "hipFree d_visited");
        }
        printf("\n\n");
    }

    return 0;
}
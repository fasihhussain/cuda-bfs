#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <queue>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

using namespace std;

namespace cg = cooperative_groups;

inline hipError_t checkCudaErr(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
    }
    return err;
}

void cpuBFS(int node, int nodes, int *adjacencyList, int *edgesOffset)
{
    queue<int> output_queue;
    vector<bool> visited(nodes, false);
    queue<int> temp_queue;

    output_queue.push(node);
    temp_queue.push(node);
    visited[node] = true;

    while (!temp_queue.empty())
    {
        int u = temp_queue.front();
        temp_queue.pop();
        for (int j = edgesOffset[u]; j < edgesOffset[u + 1]; j++)
        {
            int v = adjacencyList[j];
            if (!visited[v])
            {
                // printf("%d\n", v);
                output_queue.push(v);
                temp_queue.push(v);
                visited[v] = true;
            }
        }
    }
}

__global__ void cudaBFS(int node, int nodes, int *adjacencyList, int *edgesOffset, int *currentQueue, int *nextQueue, int *visited)
{
    int thid = threadIdx.x;
    extern __shared__ int queues[];

    int *currentQueueSize = &queues[0];
    int *nextQueueSize = currentQueueSize + 1;

    if (thid == 0)
    {
        // printf("ADJ LIST FROM GPU:\n");
        // for (int i = 0; i < edgesOffset[nodes]; i++)
        // {
        //     printf("%d ", adjacencyList[i]);
        // }
        // printf("\n\n");

        // printf("OFFSET FROM GPU:\n");
        // for (int i = 0; i < nodes + 1; i++)
        // {
        //     printf("%d ", edgesOffset[i]);
        // }
        // printf("\n\n");

        currentQueue[0] = node;
        visited[node] = 1;
        *currentQueueSize = 1;
        *nextQueueSize = 0;

        // printf("QUEUE SIZE: %d\n", *currentQueueSize);
        // for (int i = 0; i < *currentQueueSize; i++)
        // {
        //     printf("%d ", currentQueue[i]);
        // }
        // printf("\n");
    }

    __syncthreads();

    while (*currentQueueSize > 0)
    {
        for (int i = thid; i < *currentQueueSize; i += blockDim.x)
        {
            int u = currentQueue[i];
            // printf("VISITED[%d] = %d\n", u, visited[u]);
            for (int j = edgesOffset[u]; j < edgesOffset[u + 1]; j++)
            {
                int v = adjacencyList[j];
                // printf("u = %d, v = %d, VISITED = %d\n", u, v, visited[v]);
                if (visited[v] == 0)
                {
                    visited[v] = 1;
                    int position = atomicAdd(nextQueueSize, 1);
                    nextQueue[position] = v;
                }
            }
        }
        __syncthreads();

        // if (thid == 0)
        // {
        //     printf("NEXT QUEUE SIZE: %d\n", *nextQueueSize);
        //     for (int i = 0; i < *nextQueueSize; i++)
        //     {
        //         printf("%d ", nextQueue[i]);
        //     }
        //     printf("\n\n\n");
        // }
        int *temp = currentQueue;
        currentQueue = nextQueue;
        nextQueue = temp;

        temp = currentQueueSize;
        currentQueueSize = nextQueueSize;
        nextQueueSize = temp;

        if (thid == 0)
        {
            *nextQueueSize = 0;
        }
        __syncthreads();
    }
}

__global__ void updatedCudaBFS(int node, int nodes, int *adjacencyList, int *edgesOffset, int *currentQueue, int *nextQueue, int *currentQueueSize, int *nextQueueSize, int *visited)
{
    int thid = threadIdx.x;
    int blkid = blockIdx.x;

    auto g = cg::this_grid();

    if (blkid == 0 && thid == 0)
    {
        currentQueue[0] = node;
        visited[node] = 1;
        *currentQueueSize = 1;
        *nextQueueSize = 0;
    }

    g.sync();

    while (*currentQueueSize > 0)
    {
        for (int i = blkid; i < *currentQueueSize; i += gridDim.x)
        {
            int u = currentQueue[i];
            for (int j = thid; j < edgesOffset[u + 1]; j += blockDim.x)
            {
                int v = adjacencyList[j];

                if (visited[v] == 0)
                {
                    visited[v] = 1;
                    int position = atomicAdd(nextQueueSize, 1);
                    nextQueue[position] = v;
                }
            }
        }
        g.sync();

        int *temp = currentQueue;
        currentQueue = nextQueue;
        nextQueue = temp;

        temp = currentQueueSize;
        currentQueueSize = nextQueueSize;
        nextQueueSize = temp;

        if (blkid == 0 && thid == 0)
        {
            *nextQueueSize = 0;
        }
        g.sync();
    }
}

int main(void)
{
    vector<int> v_nodes = {10, 100, 500, 1000, 5000, 10000};
    vector<double> v_prob = {0.05, 0.1, 0.3, 0.5};

    for (double p : v_prob)
    {
        printf("PROBABILITY: %f\n", p);
        for (int nodes : v_nodes)
        {

            vector<int> adjacencyList;
            int *h_edgesOffset = new int[nodes + 1];

            for (int i = 0; i < nodes; i++)
            {
                h_edgesOffset[i] = adjacencyList.size();
                for (int j = 0; j < nodes; j++)
                {
                    if ((float)rand() / RAND_MAX < p)
                    {
                        if (i != j)
                        {
                            adjacencyList.push_back(j);
                        }
                    }
                }
            }
            h_edgesOffset[nodes] = adjacencyList.size();

            int *h_adjacencyList = new int[adjacencyList.size()];
            for (int i = 0; i < adjacencyList.size(); i++)
            {
                h_adjacencyList[i] = adjacencyList[i];
            }

            int *h_visited = new int[nodes];
            int *d_adjacencyList, *d_edgesOffset, *d_currentQueue, *d_nextQueue, *d_visited, *d_currentQueueSize, *d_nextQueueSize;

            checkCudaErr(hipMalloc((void **)&d_adjacencyList, adjacencyList.size() * sizeof(int)), "hipMalloc d_adjacencyList");
            checkCudaErr(hipMalloc((void **)&d_edgesOffset, (nodes + 1) * sizeof(int)), "hipMalloc d_edgesOffset");
            checkCudaErr(hipMalloc((void **)&d_currentQueue, nodes * sizeof(int)), "hipMalloc d_currentQueue");
            checkCudaErr(hipMalloc((void **)&d_nextQueue, nodes * sizeof(int)), "hipMalloc d_nextQueue");
            checkCudaErr(hipMalloc((void **)&d_visited, nodes * sizeof(int)), "hipMalloc d_visited");
            checkCudaErr(hipMalloc((void **)&d_currentQueueSize, sizeof(int)), "hipMalloc d_currentQueueSize");
            checkCudaErr(hipMalloc((void **)&d_nextQueueSize, sizeof(int)), "hipMalloc d_nextQueueSize");

            checkCudaErr(hipMemcpy(d_adjacencyList, h_adjacencyList, adjacencyList.size() * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy d_adjacencyList");
            checkCudaErr(hipMemcpy(d_edgesOffset, h_edgesOffset, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy d_edgesOffset");

            int NUMBER_OF_BLOCKS = 128;
            int NUMBER_OF_THREADS = 32;
            int SHARED_MEMORY_SIZE = 2 * sizeof(int);

            clock_t start, end;

            double time_taken_cpu = 0;
            double time_taken_gpu = 0;
            double time_taken_gpu_updated = 0;

            double t_cpu, t_gpu, t_gpu_updated;

            int SAMPLES = 10;

            for (int node = 0; node < SAMPLES; node++)
            {
                // CPU
                start = clock();
                cpuBFS(node, nodes, h_adjacencyList, h_edgesOffset);
                end = clock();

                t_cpu = ((double)(end - start)) * 1000 / CLOCKS_PER_SEC;
                time_taken_cpu += t_cpu;

                // GPU
                hipMemset(d_visited, 0, nodes * sizeof(int));
                hipOccupancyMaxPotentialBlockSize(&NUMBER_OF_BLOCKS, &NUMBER_OF_THREADS, cudaBFS, 0, 0);

                start = clock();
                cudaBFS<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS, SHARED_MEMORY_SIZE>>>(node, nodes, d_adjacencyList, d_edgesOffset, d_currentQueue, d_nextQueue, d_visited);
                checkCudaErr(hipDeviceSynchronize(), "device synchronize");
                end = clock();

                t_gpu = ((double)(end - start)) * 1000 / CLOCKS_PER_SEC;
                time_taken_gpu += t_gpu;

                hipMemcpy(h_visited, d_visited, nodes * sizeof(int), hipMemcpyDeviceToHost);

                // GPU Updated
                checkCudaErr(hipMemset(d_visited, 0, nodes * sizeof(int)), "hipMemset d_visited");
                checkCudaErr(hipMemset(d_currentQueueSize, 0, sizeof(int)), "hipMemset d_currentQueueSize");
                checkCudaErr(hipMemset(d_nextQueueSize, 0, sizeof(int)), "hipMemset d_nextQueueSize");
                hipOccupancyMaxPotentialBlockSize(&NUMBER_OF_BLOCKS, &NUMBER_OF_THREADS, updatedCudaBFS, 0, 0);

                start = clock();
                updatedCudaBFS<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS>>>(node, nodes, d_adjacencyList, d_edgesOffset, d_currentQueue, d_nextQueue, d_currentQueueSize, d_nextQueueSize, d_visited);
                checkCudaErr(hipDeviceSynchronize(), "device synchronize");
                end = clock();

                t_gpu_updated = ((double)(end - start)) * 1000 / CLOCKS_PER_SEC;
                time_taken_gpu_updated += t_gpu_updated;

                hipMemcpy(h_visited, d_visited, nodes * sizeof(int), hipMemcpyDeviceToHost);
            }

            time_taken_cpu /= SAMPLES;
            time_taken_gpu /= SAMPLES;
            time_taken_gpu_updated /= SAMPLES;

            printf("NODES: %d\t\tCPU: %f ms\t\tGPU: %f ms\t\tGPU Updated: %f ms\n", nodes, time_taken_cpu, time_taken_gpu, time_taken_gpu_updated);

            free(h_adjacencyList);
            free(h_edgesOffset);
            free(h_visited);

            checkCudaErr(hipFree(d_adjacencyList), "hipFree d_adjacencyList");
            checkCudaErr(hipFree(d_edgesOffset), "hipFree d_edgesOffset");
            checkCudaErr(hipFree(d_currentQueue), "hipFree d_currentQueue");
            checkCudaErr(hipFree(d_nextQueue), "hipFree d_nextQueue");
            checkCudaErr(hipFree(d_visited), "hipFree d_visited");
            checkCudaErr(hipFree(d_currentQueueSize), "hipFree d_currentQueueSize");
            checkCudaErr(hipFree(d_nextQueueSize), "hipFree d_nextQueueSize");
        }
        printf("\n\n");
    }

    return 0;
}

// nvcc -L /usr/lib/x86_64-linux-gnu --std=c++11 -rdc=true -gencode=arch=compute_75,code=sm_75 updated.cu -o updated.out